#include "hip/hip_runtime.h"

#include "CUDAWorker.cuh"


#include <stdio.h>
#include <iostream>

#include <time.h> // time seed
#include <stdlib.h>     /* srand, rand */
#include <time.h> 


#include "StuffWorker.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__device__
A3DVectorGPU operator+(const A3DVectorGPU& p, const A3DVectorGPU& v)
{
	return A3DVectorGPU(p._x + v._x, p._y + v._y, p._z + v._z);
}

__device__
A3DVectorGPU operator-(const A3DVectorGPU& p, const A3DVectorGPU& v)
{
	return A3DVectorGPU(p._x - v._x, p._y - v._y, p._z - v._z);
}

__device__
A3DVectorGPU operator*(const A3DVectorGPU& p, const float& f)
{
	return A3DVectorGPU(p._x * f, p._y * f, p._z * f);
}

__device__
A3DVectorGPU operator/(const A3DVectorGPU& p, const float& f)
{
	return A3DVectorGPU(p._x / f, p._y / f, p._z / f);
}


// length of a vector
__device__
float Length(const A3DVectorGPU& p) {
	return sqrt(p._x * p._x +
		p._y * p._y +
		p._z * p._z);
}

__device__
A3DVectorGPU Norm(const A3DVectorGPU& p) // get the unit vector
{
	float vlength = sqrt(p._x * p._x + p._y * p._y + p._z * p._z);

	if (vlength == 0) { return A3DVectorGPU(0, 0, 0); }

	return A3DVectorGPU(p._x / vlength,
		p._y / vlength,
		p._z / vlength);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void Simulate_GPU(A3DVectorGPU* pos_array, 
	                     A3DVectorGPU* velocity_array, 
	                     A3DVectorGPU* edge_force_array,
						 A3DVectorGPU* z_force_array,
						 A3DVectorGPU* repulsion_force_array,
						 A3DVectorGPU* boundary_force_array,
						 A3DVectorGPU* overlap_force_array,
						 A3DVectorGPU* rotation_force_array, 
						 int n, 
						 float dt, 
	                     float velocity_cap_dt)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		// oiler
		velocity_array[i] = velocity_array[i] +

			                ((edge_force_array[i] +
							z_force_array[i] +
							repulsion_force_array[i] +
							boundary_force_array[i] +
							overlap_force_array[i] +
							rotation_force_array[i]) * dt);

		float len = Length(velocity_array[i]);

		if (len > velocity_cap_dt)
		{
			velocity_array[i] = Norm(velocity_array[i]) * velocity_cap_dt;
		}

		pos_array[i] = pos_array[i] + velocity_array[i] * dt;
	}
}

void CUDAWorker::Simulate(float dt, float velocity_cap)
{
	int blockSize = SystemParams::_cuda_block_size;
	int numBlocks = (_num_vertex + blockSize - 1) / blockSize;
	Simulate_GPU <<<numBlocks, blockSize >>> (_pos_array,
		_velocity_array,
		_edge_force_array,
		_z_force_array,
		_repulsion_force_array,
		_boundary_force_array,
		_overlap_force_array,
		_rotation_force_array,
		_num_vertex,
		dt,
		velocity_cap * dt);

	hipDeviceSynchronize();
}

CUDAWorker::CUDAWorker()
{
	//TestCUDA();
	int nDevices;

	std::cout << "===== CUDA =====\n";
	
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) 
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		int maxThreadPerBlock = prop.maxThreadsPerBlock;
		int maxGridSize = prop.maxGridSize[0];
		
		std::cout << "  maxThreadsPerBlock: " << maxThreadPerBlock <<"\n";
		std::cout << "  maxGridSize: " << maxGridSize << "\n";
	}
	std::cout << "================\n";

	/*
	_edge_force_array = 0;
	_z_force_array = 0;
	_repulsion_force_array = 0;
	_boundary_force_array = 0;
	_overlap_force_array = 0;
	_rotation_force_array = 0;
	*/
}

CUDAWorker::~CUDAWorker()
{
	hipFree(_edge_force_array);
	hipFree(_z_force_array);
	hipFree(_repulsion_force_array);
	hipFree(_boundary_force_array);
	hipFree(_overlap_force_array);
	hipFree(_rotation_force_array);
}

void CUDAWorker::InitCUDA(int num_vertex)
{
	_num_vertex = num_vertex;

	// mass positions
	hipMallocManaged(&_pos_array, num_vertex * sizeof(A3DVectorGPU));

	// mass velocities
	hipMallocManaged(&_velocity_array, num_vertex * sizeof(A3DVectorGPU));
	
	// mass forces
	hipMallocManaged(&_edge_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_z_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_repulsion_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_boundary_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_overlap_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_rotation_force_array, num_vertex * sizeof(A3DVectorGPU));

	/*for (unsigned int a = 0; a < _num_vertex; a++)
	{
		_edge_force_array[a] = A3DVectorGPU();
	}*/
	
}

void CUDAWorker::RetrievePositionAndVelocityData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_GPU2CPU(&_pos_array[idx], StuffWorker::_element_list[a]._massList[b]._pos);
			CopyVector_GPU2CPU(&_velocity_array[idx], StuffWorker::_element_list[a]._massList[b]._velocity);
			idx++;
		}
	}
}

void CUDAWorker::SendPositionAndVelocityData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._pos, &_pos_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._velocity, &_velocity_array[idx]);
			idx++;
		}
	}
}

void CUDAWorker::SendForceData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._edgeForce,      &_edge_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._zForce,         &_z_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._repulsionForce, &_repulsion_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._boundaryForce,  &_boundary_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._overlapForce,   &_overlap_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._rotationForce,  &_rotation_force_array[idx]);

			idx++;
		}
	}

	// test, delete me
	/*idx = 0;
	for (unsigned int a = 0; a < _num_vertex; a++)
	{
		std::cout << _edge_force_array[a]._x << ", " << _edge_force_array[a]._y << ", " << _edge_force_array[a]._z << "\n";
	}

	std::cout << "done\n";*/
}



void CUDAWorker::CopyVector_CPU2GPU(const A3DVector& src, A3DVectorGPU* dest)
{
	dest->_x = src._x;
	dest->_y = src._y;
	dest->_z = src._z;
}

void CUDAWorker::CopyVector_GPU2CPU(A3DVectorGPU* src, A3DVector& dest)
{
	dest._x = src->_x;
	dest._y = src->_y;
	dest._z = src->_z;
}

int CUDAWorker::TestCUDA()
{
	/*const int arraySize = 20;
	const int a[arraySize] = { 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,  1, 2, 3, 4, 5, 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };*/

	/*const int arraySize = 1000;
	int a[arraySize] = { 0 };
	int b[arraySize] = { 0 };
	int c[arraySize] = { 0 };
	*/

	// dynamic
	int arraySize = 1024;
	int* a = new int[arraySize];
	int* b = new int[arraySize];
	int* c = new int[arraySize];
	   
	for (int i = 0; i < arraySize; i++)
	{
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	//std::cout << "test\n";

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	/*for (int i = 0; i < arraySize; i++)
	{
		std::cout << i << " --> " << a[i] << " + " << b[i] << " = " << c[i] << "\n";
	}*/


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	delete[] a;
	delete[] b;
	delete[] c;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "--> addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}