#include "hip/hip_runtime.h"

#include "CUDAWorker.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <time.h> // time seed
#include <stdlib.h>     /* srand, rand */
#include <time.h> 

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

CUDAWorker::CUDAWorker()
{
	//TestCUDA();
	int nDevices;

	std::cout << "===== CUDA =====\n";
	
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) 
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		int maxThreadPerBlock = prop.maxThreadsPerBlock;
		int maxGridSize = prop.maxGridSize[0];
		
		std::cout << "  maxThreadsPerBlock: " << maxThreadPerBlock <<"\n";
		std::cout << "  maxGridSize: " << maxGridSize << "\n";
	}
	std::cout << "================\n";
}

CUDAWorker::~CUDAWorker()
{

}

int CUDAWorker::TestCUDA()
{
	/*const int arraySize = 20;
	const int a[arraySize] = { 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,  1, 2, 3, 4, 5, 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };*/

	/*const int arraySize = 1000;
	int a[arraySize] = { 0 };
	int b[arraySize] = { 0 };
	int c[arraySize] = { 0 };
	*/

	// dynamic
	int arraySize = 1024;
	int* a = new int[arraySize];
	int* b = new int[arraySize];
	int* c = new int[arraySize];
	   
	for (int i = 0; i < arraySize; i++)
	{
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	//std::cout << "test\n";

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	/*for (int i = 0; i < arraySize; i++)
	{
		std::cout << i << " --> " << a[i] << " + " << b[i] << " = " << c[i] << "\n";
	}*/


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	delete[] a;
	delete[] b;
	delete[] c;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "--> addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}