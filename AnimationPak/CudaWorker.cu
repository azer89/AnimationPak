#include "hip/hip_runtime.h"

#include "CUDAWorker.cuh"


#include <stdio.h>
#include <iostream>

#include <time.h> // time seed
#include <stdlib.h>     /* srand, rand */
#include <time.h> 


#include "StuffWorker.h"

__device__
A3DVectorGPU operator+(const A3DVectorGPU& p, const A3DVectorGPU& v)
{
	return A3DVectorGPU(p._x + v._x, p._y + v._y, p._z + v._z);
}

__device__
A3DVectorGPU operator-(const A3DVectorGPU& p, const A3DVectorGPU& v)
{
	return A3DVectorGPU(p._x - v._x, p._y - v._y, p._z - v._z);
}

__device__
A3DVectorGPU operator*(const A3DVectorGPU& p, const float& f)
{
	return A3DVectorGPU(p._x * f, p._y * f, p._z * f);
}

__device__
A3DVectorGPU operator/(const A3DVectorGPU& p, const float& f)
{
	return A3DVectorGPU(p._x / f, p._y / f, p._z / f);
}


// length of a vector
__device__
float Length(const A3DVectorGPU& p) {
	return sqrt(p._x * p._x +
		p._y * p._y +
		p._z * p._z);
}

__device__
A3DVectorGPU Norm(const A3DVectorGPU& p) // get the unit vector
{
	float vlength = sqrt(p._x * p._x + p._y * p._y + p._z * p._z);

	if (vlength == 0) { return A3DVectorGPU(0, 0, 0); }

	return A3DVectorGPU(p._x / vlength,
		p._y / vlength,
		p._z / vlength);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void Simulate_GPU(A3DVectorGPU* pos_array, 
	                     A3DVectorGPU* velocity_array, 
	                     A3DVectorGPU* edge_force_array,
						 A3DVectorGPU* z_force_array,
						 A3DVectorGPU* repulsion_force_array,
						 A3DVectorGPU* boundary_force_array,
						 A3DVectorGPU* overlap_force_array,
						 A3DVectorGPU* rotation_force_array, 
						 int n, 
						 float dt, 
	                     float velocity_cap_dt)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		// oiler
		velocity_array[i] = velocity_array[i] +

			                ((edge_force_array[i] +
							z_force_array[i] +
							repulsion_force_array[i] +
							boundary_force_array[i] +
							overlap_force_array[i] +
							rotation_force_array[i]) * dt);

		float len = Length(velocity_array[i]);

		if (len > velocity_cap_dt)
		{
			velocity_array[i] = Norm(velocity_array[i]) * velocity_cap_dt;
		}

		pos_array[i] = pos_array[i] + velocity_array[i] * dt;
	}
}

void CUDAWorker::Simulate(float dt, float velocity_cap)
{
	int blockSize = SystemParams::_cuda_block_size;
	int numBlocks = (_num_vertex + blockSize - 1) / blockSize;
	Simulate_GPU <<<numBlocks, blockSize >>> (_pos_array,
		_velocity_array,
		_edge_force_array,
		_z_force_array,
		_repulsion_force_array,
		_boundary_force_array,
		_overlap_force_array,
		_rotation_force_array,
		_num_vertex,
		dt,
		velocity_cap * dt);

	hipDeviceSynchronize();
}

CUDAWorker::CUDAWorker()
{
	//TestCUDA();
	int nDevices;

	std::cout << "===== CUDA =====\n";
	
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) 
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		int maxThreadPerBlock = prop.maxThreadsPerBlock;
		int maxGridSize = prop.maxGridSize[0];
		
		std::cout << "  maxThreadsPerBlock: " << maxThreadPerBlock <<"\n";
		std::cout << "  maxGridSize: " << maxGridSize << "\n";
	}
	std::cout << "================\n";

	/*
	_edge_force_array = 0;
	_z_force_array = 0;
	_repulsion_force_array = 0;
	_boundary_force_array = 0;
	_overlap_force_array = 0;
	_rotation_force_array = 0;
	*/
}

CUDAWorker::~CUDAWorker()
{
	hipFree(_edge_force_array);
	hipFree(_z_force_array);
	hipFree(_repulsion_force_array);
	hipFree(_boundary_force_array);
	hipFree(_overlap_force_array);
	hipFree(_rotation_force_array);

	hipFree(_pos_array);
	hipFree(_velocity_array);
	hipFree(_spring_array);
	hipFree(_spring_parameters);
}

void CUDAWorker::InitCUDA(int num_vertex, int num_spring)
{
	_num_vertex = num_vertex;
	_num_spring = num_spring;

	// mass positions
	hipMallocManaged(&_pos_array, num_vertex * sizeof(A3DVectorGPU));

	// mass velocities
	hipMallocManaged(&_velocity_array, num_vertex * sizeof(A3DVectorGPU));
	
	// mass forces
	hipMallocManaged(&_edge_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_z_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_repulsion_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_boundary_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_overlap_force_array, num_vertex * sizeof(A3DVectorGPU));
	hipMallocManaged(&_rotation_force_array, num_vertex * sizeof(A3DVectorGPU));

	// springs
	hipMallocManaged(&_spring_array, num_spring * sizeof(A3DVectorGPU));

	// spring parameters
	hipMallocManaged(&_spring_parameters, 4 * sizeof(float));
	_spring_parameters[0] = SystemParams::_k_edge;
	_spring_parameters[1] = SystemParams::_k_time_edge;
	_spring_parameters[2] = SystemParams::_k_edge;
	_spring_parameters[3] = SystemParams::_k_neg_space_edge;
	
}

void CUDAWorker::RetrievePositionAndVelocityData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_GPU2CPU(&_pos_array[idx], StuffWorker::_element_list[a]._massList[b]._pos);
			CopyVector_GPU2CPU(&_velocity_array[idx], StuffWorker::_element_list[a]._massList[b]._velocity);
			idx++;
		}
	}
}

void CUDAWorker::SendSpringData()
{
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._layer_springs.size(); b++)
		{
		}

		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._time_springs.size(); b++)
		{
		}

		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._auxiliary_springs.size(); b++)
		{
		}

		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._neg_space_springs.size(); b++)
		{
		}
	}
}

void CUDAWorker::SendPositionAndVelocityData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._pos, &_pos_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._velocity, &_velocity_array[idx]);
			idx++;
		}
	}
}

void CUDAWorker::SendForceData()
{
	int idx = 0;
	for (unsigned int a = 0; a < StuffWorker::_element_list.size(); a++)
	{
		for (unsigned int b = 0; b < StuffWorker::_element_list[a]._massList.size(); b++)
		{
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._edgeForce,      &_edge_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._zForce,         &_z_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._repulsionForce, &_repulsion_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._boundaryForce,  &_boundary_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._overlapForce,   &_overlap_force_array[idx]);
			CopyVector_CPU2GPU(StuffWorker::_element_list[a]._massList[b]._rotationForce,  &_rotation_force_array[idx]);

			idx++;
		}
	}

	// test, delete me
	/*idx = 0;
	for (unsigned int a = 0; a < _num_vertex; a++)
	{
		std::cout << _edge_force_array[a]._x << ", " << _edge_force_array[a]._y << ", " << _edge_force_array[a]._z << "\n";
	}

	std::cout << "done\n";*/
}



void CUDAWorker::CopyVector_CPU2GPU(const A3DVector& src, A3DVectorGPU* dest)
{
	dest->_x = src._x;
	dest->_y = src._y;
	dest->_z = src._z;
}

void CUDAWorker::CopyVector_GPU2CPU(A3DVectorGPU* src, A3DVector& dest)
{
	dest._x = src->_x;
	dest._y = src->_y;
	dest._z = src->_z;
}